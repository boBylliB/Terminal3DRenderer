#include "hip/hip_runtime.h"
#ifndef CUDACAMERA_CU
#define CUDACAMERA_CU

#include <thread>

#include "CUDACamera.cuh"

using namespace std;

// CUDA kernel functions to run in parallel on the GPU (cannot be a member function)
__global__ void CUDACalculateIntersectDistances(int* numRays, double* distances, Point* pos, Triangle* tris, int* numTris, Vector* rays) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < *numRays; idx += stride) {
		distances[idx] = CUDACalculateIntersectDistance(tris, *numTris, *pos, rays[idx]);
	}
}
__global__ void CUDACalculateIntersectDistancesSegmented(int* numRays, double* distances, Point* pos, Triangle* tris, Triangle** triSegments, int* numTris, int* numTrisPerSegment, Vector* rays, int* numSegments, bool** segmentAssignment) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < *numRays; idx += stride) {
		// Get the chunk index that the current ray resides in
		// Grab the assigned segments based on that chunk index
		// Calculate intersect distances for each segment
		// Set the resulting distance to the closest calculated distance
		distances[idx] = CUDACalculateIntersectDistance(tris, *numTris, *pos, rays[idx]);
	}
}
__global__ void CUDACalculateRays(int* outDim, Angle* outAngles, int* numRays, Vector* rays) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < *numRays; idx += stride) {
		Angle rayAngle = outAngles[0];
		rayAngle.theta += (idx % outDim[1]) * outAngles[1].theta;
		rayAngle.phi += (idx / outDim[0]) * outAngles[1].phi;

		Vector ray = CUDAAngleVector(rays[0], rayAngle);
		rays[idx] = ray;
	}
}

// CUDA device functions
__device__ double CUDACalculateIntersectDistance(Triangle* tris, int numTris, Point origin, Vector ray) {
	// Get the normal of the triangle
	// Get the vector from any point on the triangle to the origin
	// Distance = (normal dot (triangle to origin)) / (normal dot normalized ray)
	// If the distance is negative, it is behind the origin of the ray
	// Find the point of intersection by adding distance * normalized ray to the origin coordinates
	// Check for intersection within triangle
	double rayDistance = NPP_MAXABS_64F;
	bool set = false;
	Vector normRay = CUDANormalize(ray);

	for (int triIdx = 0; triIdx < numTris; triIdx++) {
		Vector originVector = CUDADifferenceVector(ray, origin, tris[triIdx].verts[0]);
		Vector normal = tris[triIdx].normal;

		double dist = CUDADot(normal, originVector) / CUDADot(normal, normRay);
		if (dist > 0 && dist < rayDistance && CUDACheckWithin(tris[triIdx], normRay, origin)) {
			set = true;
			rayDistance = dist;
		}
	}
	if (set) {
		return rayDistance;
	}
	else {
		return -1;
	}
}
__device__ bool CUDACheckWithin(Triangle tri, Vector dir, Point origin) {
	// printf("%d: I=%lf, J=%lf, K=%lf, I=%lf, J=%lf, K=%lf\n", triIdx, dir.I, dir.J, dir.K, tri.normal.I, tri.normal.J, tri.normal.K);

	// Vectors from ray origin to each vertex as the bounds
	Vector limitA = CUDADifferenceVector(dir, origin, tri.verts[0]);
	Vector limitB = CUDADifferenceVector(dir, origin, tri.verts[1]);
	Vector limitC = CUDADifferenceVector(dir, origin, tri.verts[2]);

	// Create limiting planes using the bounding vectors
	Vector planeA = CUDACross(limitB, limitC);
	// If the tested vector is on the same side of each plane as the one bounding vector not within the test plane
	// Therefore, the only way that it could be on the "inside" of each plane is if the tested vector is between the bounding vectors
	if (CUDADot(limitA, planeA) * CUDADot(dir, planeA) <= 0) return false;

	Vector planeB = CUDACross(limitA, limitC);
	if (CUDADot(limitB, planeB) * CUDADot(dir, planeB) <= 0) return false;

	Vector planeC = CUDACross(limitA, limitB);
	if (CUDADot(limitC, planeC) * CUDADot(dir, planeC) <= 0) return false;

	// If we get here, then all tests passed
	return true;
}
__device__ Vector CUDADifferenceVector(Vector vec, Point a, Point b) {
	Vector out = vec;
	out.I = b.x - a.x;
	out.J = b.y - a.y;
	out.K = b.z - a.z;
	return out;
}
__device__ Vector CUDAAngleVector(Vector vec, Angle ang) {
	Vector out = vec;
	double degToRadCoeff = (1 / 180.0)* atan(1.0) * 4.0;
	out.I = cos(degToRadCoeff*ang.theta) * sin(degToRadCoeff*ang.phi);
	out.J = cos(degToRadCoeff*ang.phi);
	out.K = sin(degToRadCoeff*ang.theta) * sin(degToRadCoeff*ang.phi);

	// If there's no way to tell how large a vector should be, I just normalize it so that it's easier to scale later
	out = CUDANormalize(out);
}
__device__ Vector CUDANormalize(Vector vec) {
	double mag = sqrt(vec.I * vec.I + vec.J * vec.J + vec.K * vec.K);
	vec.I = vec.I / mag;
	vec.J = vec.J / mag;
	vec.K = vec.K / mag;
	return vec;
}
__device__ double CUDADot(Vector a, Vector b) {
	double output = a.I * b.I + a.J * b.J + a.K * b.K;
	return output;
}
__device__ Vector CUDACross(Vector a, Vector b) {
	Vector out = a;
	out.I = a.J * b.K - a.K * b.J;
	out.J = a.K * b.I - a.I * b.K;
	out.K = a.I * b.J - a.J * b.I;
	return out;
}

CUDACamera::CUDACamera(const Point& position, const Vector& direction, const double fov = FOV, const double roll = 0, const int outHeight = HEIGHT, const int outWidth = WIDTH) : Camera(position, direction, fov, roll, outHeight, outWidth) {

}
CUDACamera::CUDACamera(Camera& cam) : Camera(cam.getPosition(), cam.getDirection(), cam.getFOV().theta, cam.getRoll(), cam.getOutputSize()[0], cam.getOutputSize()[1]) {

}

// Core Functions
// Functions the same as the standard camera display, but does the math in parallel on the GPU for speed
void CUDACamera::CUDADisplay(const Mesh& m, const bool dither) {
	CUDADisplayMath(m, dither).print();
}
// Just the math of the display function above, outputting to a Frame to be displayed later
Frame CUDACamera::CUDADisplayMath(const Mesh& m, const bool dither) {
	// Initializing profiler
	Profiler profiler;

	// Display a line for the output width for verification that the whole display fits on screen
	for (int idx = 0; idx < outputWidth; idx++) {
		cout << "@";
	}
	cout << endl << endl;
	// Calculate the angle between pixels
	Angle angleBetween(fieldOfView.theta / outputWidth, fieldOfView.phi / outputHeight);
	Angle startingAngle((angleBetween.theta * (outputWidth / 2.0) * -1.0), (angleBetween.phi * (outputHeight / 2.0) * -1.0));
	startingAngle += direction.toAngle();
	// Generate the angles to split at for each ray chunk
	Angle splitAngle(fieldOfView.theta / (NUMCHUNKSX + 1), fieldOfView.phi / (NUMCHUNKSY + 1));
	// Create Vectors to form the bounding planes
	vector<vector<Vector>> boundingVectors;
	for (int idx = 0; idx < NUMCHUNKSX + 2; idx++) {
		vector<Vector> currentCol;
		for (int jdx = 0; jdx < NUMCHUNKSY + 2; jdx++) {
			Angle currentAngle(startingAngle.theta + idx * splitAngle.theta, startingAngle.phi + jdx * splitAngle.phi);
			Vector currentVec(currentAngle);
			currentCol.push_back(currentVec);
		}
		boundingVectors.push_back(currentCol);
	}
	profiler.start("Segment Allocation");
	// Segments may lie on both sides of a bounding plane, in which case it should be assigned to all affected chunks
	vector<vector<bool>> segmentsPerChunk; // each boolean represents a segment, each vector of booleans represents the segments assigned to a chunk of rays
	// We have to initialize these to zero for future math to work, even though we won't be directly using these on the first pass
	Vector boundingPlaneY[2] = {};
	Vector boundingPlaneX[2] = {};
	for (int idx = 0; idx < NUMCHUNKSX + 1; idx++) {
		for (int jdx = 0; jdx < NUMCHUNKSY + 1; jdx++) {
			// Generate bounding planes
			boundingPlaneX[1] = boundingPlaneX[0];
			boundingPlaneY[0] = boundingVectors[idx + 1][jdx + 1].cross(boundingVectors[idx + 2][jdx + 1]);
			boundingPlaneX[1] = boundingPlaneX[0];
			boundingPlaneX[0] = boundingVectors[idx + 1][jdx + 1].cross(boundingVectors[idx + 1][jdx]);
			// We move from low angle to high angle, so each iteration should pull chunks from the lower side of the plane
			// If we're at an iteration greater than 0, we should also take into account the previous split, ignoring any chunks completely covered by previous planes
			vector<bool> pickedSegments;
			for (int kdx = 0; kdx < m.numSegments; kdx++) {
				// Compare the segment center against the bounding planes using a Vector to the top left corner as reference
				Vector toCenter(position, m.segmentCenters[kdx]);
				if (boundingVectors[0][0].dot(boundingPlaneX[0]) * toCenter.dot(boundingPlaneX[0]) >= 0 && boundingVectors[0][0].dot(boundingPlaneY[0]) * toCenter.dot(boundingPlaneY[0]) >= 0) {
					// Make sure at least one of the vertices of the segment bounding box is on the same side of the previous bounding planes as the bottom right corner
					bool notExcluded = false;
					for (int xShift = -1; xShift < 2; xShift += 2) {
						for (int yShift = -1; yShift < 2; yShift += 2) {
							for (int zShift = -1; zShift < 2; zShift += 2) {
								Point corner = m.segmentCenters[kdx];
								corner.x += m.segmentBounds[kdx].I * xShift;
								corner.y += m.segmentBounds[kdx].J * yShift;
								corner.z += m.segmentBounds[kdx].K * zShift;
								Vector toCorner(position, corner);
								// If this is iteration 0, there are no previous bounding planes, so it cannot have been excluded
								if ((boundingVectors[NUMCHUNKSX + 1][NUMCHUNKSY + 1].dot(boundingPlaneX[1]) * toCorner.dot(boundingPlaneX[1]) >= 0 || idx == 0)
									&& (boundingVectors[NUMCHUNKSX + 1][NUMCHUNKSY + 1].dot(boundingPlaneY[1]) * toCenter.dot(boundingPlaneY[1]) >= 0 || jdx == 0)) {
									notExcluded = true;
								}
							}
						}
					}
					// If true, add it to be picked and continue
					if (notExcluded) {
						pickedSegments.push_back(true);
						continue;
					}
				}
				else {
					// Check if at least one of the vertices of the segment bounding box is on the same side of the current bounding planes as the top left corner
					bool notExcluded = false;
					for (int xShift = -1; xShift < 2; xShift += 2) {
						for (int yShift = -1; yShift < 2; yShift += 2) {
							for (int zShift = -1; zShift < 2; zShift += 2) {
								Point corner = m.segmentCenters[kdx];
								corner.x += m.segmentBounds[kdx].I * xShift;
								corner.y += m.segmentBounds[kdx].J * yShift;
								corner.z += m.segmentBounds[kdx].K * zShift;
								Vector toCorner(position, corner);
								if (boundingVectors[0][0].dot(boundingPlaneX[0]) * toCorner.dot(boundingPlaneX[0]) >= 0
									&& boundingVectors[0][0].dot(boundingPlaneY[0]) * toCenter.dot(boundingPlaneY[0]) >= 0) {
									notExcluded = true;
								}
							}
						}
					}
					// If true, add it to be picked and continue
					if (notExcluded) {
						pickedSegments.push_back(true);
						continue;
					}
				}
				// If nothing passes, add false to the vector
				pickedSegments.push_back(false);
			}
			segmentsPerChunk.push_back(pickedSegments);
		}
	}
	profiler.end();
	profiler.start("Memory Allocation");
	// Create required arrays & data on shared GPU & CPU memory
	int* numRays;
	hipMallocManaged(&numRays, sizeof(int));
	*numRays = outputHeight * outputWidth;
	Point* pos;
	hipMallocManaged(&pos, sizeof(Point));
	*pos = position;
	int* numTris;
	hipMallocManaged(&numTris, sizeof(int));
	*numTris = m.tris.size();
	double* intersectDistances;
	hipMallocManaged(&intersectDistances, *numRays * sizeof(double));
	Triangle* triArr;
	hipMallocManaged(&triArr, *numTris * sizeof(Triangle));
	for (int idx = 0; idx < *numTris; idx++) {
		triArr[idx] = m.tris[idx];
	}
	int* outDim;
	hipMallocManaged(&outDim, 2 * sizeof(int));
	outDim[0] = outputHeight;
	outDim[1] = outputWidth;
	Angle* outAngles;
	hipMallocManaged(&outAngles, 2 * sizeof(Angle));
	outAngles[0] = startingAngle;
	outAngles[1] = angleBetween;
	int* numSegments;
	hipMallocManaged(&numSegments, sizeof(int));
	*numSegments = m.numSegments;
	// Create rays array on shared CPU & GPU memory
	vector<Vector> rayVector;
	Vector* rays;
	hipMallocManaged(&rays, *numRays * sizeof(Vector));
	// Segment assignment is an array of booleans for each ray that is read for each ray to determine which tri segments to check for intersections
	bool** segmentAssignment;
	hipMallocManaged(&segmentAssignment, *numRays * (*numSegments * sizeof(bool)));
	profiler.end();
	profiler.start("Ray Calculation");
	vector<vector<bool>> segmentAssignmentVector;
	for (int row = 0; row < outputHeight; row++) {
		for (int col = 0; col < outputWidth; col++) {
			Angle rayAngle = startingAngle;
			rayAngle.theta += col * angleBetween.theta;
			rayAngle.phi += row * angleBetween.phi;

			Vector ray(rayAngle);
			rayVector.push_back(ray);

			// Determine which chunk this ray belongs to
			Vector boundingPlaneY[2] = {};
			Vector boundingPlaneX[2] = {};
			for (int idx = 0; idx < NUMCHUNKSX + 1; idx++) {
				for (int jdx = 0; jdx < NUMCHUNKSY + 1; jdx++) {
					// Generate bounding planes
					boundingPlaneX[1] = boundingPlaneX[0];
					boundingPlaneY[0] = boundingVectors[idx + 1][jdx + 1].cross(boundingVectors[idx + 2][jdx + 1]);
					boundingPlaneX[1] = boundingPlaneX[0];
					boundingPlaneX[0] = boundingVectors[idx + 1][jdx + 1].cross(boundingVectors[idx + 1][jdx]);
					// Test that this vector lies on the same side of the current bounding planes as the top left corner
					// And that this vector lies on the same side of the previous bounding planes as the bottom right corner
					// (If this is the first bounding plane for either index, this must be true but would normally return false)
					if (boundingVectors[0][0].dot(boundingPlaneX[0]) * ray.dot(boundingPlaneX[0]) >= 0
						&& boundingVectors[0][0].dot(boundingPlaneY[0]) * ray.dot(boundingPlaneY[0]) >= 0
						&& (boundingVectors[NUMCHUNKSX + 1][NUMCHUNKSY + 1].dot(boundingPlaneX[1]) * ray.dot(boundingPlaneX[1]) >= 0 || idx == 0)
						&& (boundingVectors[NUMCHUNKSX + 1][NUMCHUNKSY + 1].dot(boundingPlaneY[1]) * ray.dot(boundingPlaneY[1]) >= 0 || jdx == 0)) {
						// If all of these tests pass, this ray exists within this chunk, and we assign it the segments that its chunk needs to test
						segmentAssignmentVector.push_back(segmentsPerChunk[idx * (NUMCHUNKSY + 1) + jdx]);
					}
				}
			}
		}
	}
	profiler.end();
	for (int idx = 0; idx < *numRays; idx++) {
		rays[idx] = rayVector[idx];
		for (int jdx = 0; jdx < *numSegments; jdx++) {
			segmentAssignment[idx][jdx] = segmentAssignmentVector[idx][jdx];
		}
	}
	cout << "Rays created, calculating intersects" << endl;
	// Calculates the number of thread blocks needed, making sure to round up if needed
	int numBlocks = (*numRays + NUMTHREADSPERBLOCK - 1) / NUMTHREADSPERBLOCK;
	// Prefetch the needed data from the CPU onto the GPU before running the relevant GPU code
	int device = -1;
	hipGetDevice(&device);
	hipMemPrefetchAsync(numRays, sizeof(int), device, NULL);
	hipMemPrefetchAsync(intersectDistances, *numRays * sizeof(double), device, NULL);
	hipMemPrefetchAsync(pos, sizeof(Point), device, NULL);
	hipMemPrefetchAsync(triArr, *numTris * sizeof(Triangle), device, NULL);
	hipMemPrefetchAsync(numTris, sizeof(int), device, NULL);
	hipMemPrefetchAsync(rays, *numRays * sizeof(Vector), device, NULL);
	//// Calculate rays on the GPU
	//profiler.start("GPU Ray Calculation");
	//CUDACalculateRays<<<numBlocks, NUMTHREADSPERBLOCK>>>(outDim, outAngles, numRays, rays);
	//// Now we wait for all threads to finish and collect the results
	//hipDeviceSynchronize();
	//profiler.end();
	//int raysMatch = 0;
	//for (int idx = 0; idx < *numRays; idx++) {
	//	if (rays[idx] == rayVector[idx]) raysMatch++;
	//	rays[idx] = rayVector[idx];
	//}
	//cout << raysMatch << " rays matched out of " << *numRays << endl;
	// Calculate the intersection distances for each ray in parallel
	cout << "Beginning GPU Distance Calculations" << endl;
	profiler.start("GPU Distance Calculations");
	CUDACalculateIntersectDistances<<<numBlocks, NUMTHREADSPERBLOCK>>>(numRays, intersectDistances, pos, triArr, numTris, rays);
	// Now we wait for all threads to finish and collect the results
	hipDeviceSynchronize();
	//cout << "I=" << rays[0].I << ", J=" << rays[0].J << ", K=" << rays[0].K << ", I=" << triArr[0].normal.I << ", J=" << triArr[0].normal.J << ", K=" << triArr[0].normal.K << endl;
	profiler.end();
	cout << "Calculated Distances" << endl;

	profiler.start("Brightness falloff calculations");
	vector<double> distances;
	for (int idx = 0; idx < *numRays; idx++) {
		distances.push_back(intersectDistances[idx]);
	}
	// Calculate the minimum distance for brightness falloff
	Frame frame(distances, outputHeight, outputWidth, dither);
	frame.trimPixels();
	profiler.end();

	profiler.start("Freeing memory");
	// Free memory
	for (int idx = 0; idx < NUMTHREADS; idx++) {
		hipFree(numRays);
		hipFree(intersectDistances);
		hipFree(pos);
		hipFree(triArr);
		hipFree(numTris);
		hipFree(rays);
	}
	profiler.end();

	profiler.printSegments();
	profiler.archiveSegments();

	return frame;
}

#endif